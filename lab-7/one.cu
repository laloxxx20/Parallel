#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <time.h>       /* time */

using namespace std;


__global__
void vecAddKernel(float* A, float* B, float* C, int n)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if(i<n) C[i] = A[i] + B[i];
}

void print_vec(float* vector, int size)
{
    for(int i=0; i<size; i++)
        cout << vector[i] << " ";
    cout<<endl;
}

void vecAdd(float* A, float* B, float* C, int n)
{
    int size = (n * sizeof(float)) * 2;
    A = (float*)malloc(size);
    C = (float*)malloc(size);
    B = (float*)malloc(size);
    srand (time(NULL));

    for( int i = 0; i < n; i++ ) 
    {
        A[i] = rand() % n + 1;
        B[i] = rand() % n + 1;
    }

    print_vec(A, size);
    cout<<"-------------"<<endl;
    print_vec(B, size);

    float *d_A, *d_B, *d_C;

    hipMalloc((void**)&d_A, size);
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);

    hipMalloc((void**)&d_B, size);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    hipMalloc((void**)&d_C, size); 
    vecAddKernel<<<ceil((float)n/256.0), 256>>>(d_A, d_B, d_C, n);
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    hipFree(d_A); 
    hipFree(d_B); 
    hipFree(d_C);

    print_vec(C, size);
}

int main()
{
    // Size of vectors
    int n = 50;
 
    // Host input vectors
    float* h_a;
    float* h_b;
    float* h_c;

    vecAdd(h_a, h_b, h_c, n);
    return 0;
}