#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;


void initArray(float* vec, int n) {
    int i;
    for(i=0; i<n; i++)
        vec[i] = rand() % 9 + 1;
}

void initMat(float* mat, int n) {
    int i, j;
    for(i=0; i<n; i++)
        for(j=0; j<n; j++)
            mat[i*n+j] = rand() % 9 + 1;
}

void printVec(float* vector, int size)
{
    for(int i=0; i<n; i++)
        cout << vector[i] << " ";
    cout<<endl;
}

void printMat(float *a, int n) {
    int i, j;
    for(int i=0; i<n; i++){
        for (int j=0; j<n; j++)
            cout<< a[i*n+j] << " ";
    cout<<endl;
    }    
}

__global__
void mulKernel(float *vec, float *mat, float* c, int n)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x ;
    float sum=0;

    if(i < n){
        for(int j=0; j<n; j++)
            sum += vec[j]*mat[(j*n) + i];
        c[i]=sum;
    }
}

void mulVecMat(float* vec, float* mat, int n){
    float* c;
    float* dev_a, * dev_b, * dev_c;

    vec = (float*)malloc(sizeof(float)*n);
    mat = (float*)malloc(sizeof(float)*n*n);
    c = (float*)malloc(sizeof(float)*n);

    initArray(vec, n);
    initMat(mat, n*n);
    initMat(c, n);

    printVec(a, n);
    printMat(b, n);
    printVec(c, n);

    hipMalloc((void**)&dev_a, sizeof(float)*n);
    hipMemcpy(dev_a, vec, sizeof(float)*n, hipMemcpyHostToDevice);

    hipMalloc((void**)&dev_b, sizeof(float)*n*n);
    hipMemcpy(dev_b, mat, sizeof(float)*n*n, hipMemcpyHostToDevice);

    hipMalloc((void**)&dev_c, sizeof(float)*m);
    mulKernel<<<ceil(n/256.0), 256>>>(dev_a, dev_b, dev_c, n);
    hipMemcpy(c, dev_c, sizeof(float)*n, hipMemcpyDeviceToHost);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    
    printMat(c, n);
}

int main()
{
    // Size of vectors
    int n = 100;
 
    // Host input vectors
    float* h_a;
    float* h_ma;

    void mulVecMat(h_a, h_ma, n);
    return 0;
}