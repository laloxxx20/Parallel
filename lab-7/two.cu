#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;


void initArray(float* vec, int n) {
    int i;
    for(i=0; i<n; i++)
        vec[i] = rand() % 9 + 1;
}

void initMat(float* mat, int n) {
    int i, j;
    for(i=0; i<n; i++)
        for(j=0; j<n; j++)
            mat[i*n+j] = rand() % 9 + 1;
}

void printVec(float* vector, int size)
{
    for(int i=0; i<size; i++)
        cout << vector[i] << " ";
    cout<<endl;
}

void printMat(float *a, int n) {
    for(int i=0; i<n; i++){
        for (int j=0; j<n; j++)
            cout<< a[i*n+j] << " ";
    cout<<endl;
    }
}

__global__
void mulKernel(float *vec, float *mat, float* c, int n)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x ;
    float sum=0;

    if(i < n){
        for(int j=0; j<n; j++)
            sum += vec[j]*mat[(j*n) + i];
        c[i]=sum;
    }
}

void mulVecMat(float* vec, float* mat, int n){
    float* c;
    float* dev_a, * dev_b, * dev_c;
    cout<<"is oke"<<endl;

    vec = (float*)malloc(sizeof(float)*n);
    mat = (float*)malloc(sizeof(float)*n*n);
    c = (float*)malloc(sizeof(float)*n);

    cout<<"is oke"<<endl;

    initArray(vec, n);
    cout<<"array is okey"<<endl;
    initMat(mat, n*n);
    cout<<"init ,at is okey"<<endl;
    initMat(c, n);
    cout<<"init mat 2 is okey"<<endl;

    // printVec(vec, n);
    // printMat(mat, n*n);
    // printVec(c, n);

    hipMalloc((void**)&dev_a, sizeof(float)*n);
    hipMemcpy(dev_a, vec, sizeof(float)*n, hipMemcpyHostToDevice);

    hipMalloc((void**)&dev_b, sizeof(float)*n*n);
    hipMemcpy(dev_b, mat, sizeof(float)*n*n, hipMemcpyHostToDevice);

    hipMalloc((void**)&dev_c, sizeof(float)*n);
    mulKernel<<<ceil(n/256.0), 256>>>(dev_a, dev_b, dev_c, n);
    hipMemcpy(c, dev_c, sizeof(float)*n, hipMemcpyDeviceToHost);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    // printMat(c, n);
}

int main()
{
    // Size of vectors
    int n = 100;

    // Host input vectors
    float* h_a = 0;
    float* h_ma = 0;

    mulVecMat(h_a, h_ma, n);
    return 0;
}

